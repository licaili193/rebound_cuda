#include "hip/hip_runtime.h"
#include "rebound_simulation.h"
#include "rebound_gravity.h"
#include "rebound_integration.h"
#include "rebound_utils.h"
#include <iostream>
#include <cmath>
#include <cstdlib>
#include <cstring>

// ReboundCudaSimulation class implementation
ReboundCudaSimulation::ReboundCudaSimulation() {
    h_particles = nullptr;
    d_particles = nullptr;
    particles_allocated = false;
    device_particles_current = false;  // Initially device particles are not current
    particle_count = 0;
    
    // Initialize configuration with default values
    config.n_particles = 0;
    config.t = 0.0;
    config.dt = 0.01;
    config.G = 1.0;
    config.gravity_mode = GRAVITY_BASIC;
    config.softening = 0.0;
    config.opening_angle = 0.5;
    config.max_iterations = 1000000;
    config.max_tree_depth = 20;
    config.collision_detection = false;
}

ReboundCudaSimulation::~ReboundCudaSimulation() {
    // Free host memory
    if (h_particles) {
        free(h_particles);
        h_particles = nullptr;
    }
    
    // Free device memory
    if (d_particles) {
        hipFree(d_particles);
        d_particles = nullptr;
    }
    
    particles_allocated = false;
}

void ReboundCudaSimulation::initializeSimulation(int n_particles, double dt, double G) {
    // Clean up previous allocation if any
    if (h_particles) {
        free(h_particles);
        h_particles = nullptr;
    }
    if (d_particles) {
        hipFree(d_particles);
        d_particles = nullptr;
    }
    
    config.n_particles = n_particles;
    config.dt = dt;
    config.G = G;
    config.t = 0.0;
    
    // Reset particle counter for new simulation
    particle_count = 0;
    
    // Allocate host memory for particles
    h_particles = (Particle*)malloc(n_particles * sizeof(Particle));
    if (!h_particles) {
        std::cerr << "Failed to allocate host memory for particles" << std::endl;
        exit(EXIT_FAILURE);
    }
    
    memset(h_particles, 0, n_particles * sizeof(Particle));
    
    // Allocate device memory for particles
    hipError_t err = hipMalloc((void**)&d_particles, n_particles * sizeof(Particle));
    checkCudaError(err, "Failed to allocate device memory for particles");
    
    particles_allocated = true;
}

void ReboundCudaSimulation::addParticle(double m, double x, double y, double z, 
                                       double vx, double vy, double vz, double r) {
    std::cout << "DEBUG: Adding particle " << particle_count << " with mass=" << m << ", pos=(" << x << ", " << y << ", " << z << ")" << std::endl;
    
    if (particle_count >= config.n_particles) {
        std::cerr << "Error: Cannot add more particles than allocated (" << config.n_particles << ")" << std::endl;
        return;
    }
    
    if (!h_particles) {
        std::cerr << "Error: Particles not allocated. Call initializeSimulation first." << std::endl;
        return;
    }
    
    Particle& p = h_particles[particle_count];
    p.m = m;
    p.x = x; p.y = y; p.z = z;
    p.vx = vx; p.vy = vy; p.vz = vz;
    p.ax = 0.0; p.ay = 0.0; p.az = 0.0;
    p.r = r;
    
    std::cout << "DEBUG: After setting values: mass=" << p.m << ", pos=(" << p.x << ", " << p.y << ", " << p.z << ")" << std::endl;
    
    particle_count++;
    device_particles_current = false;  // Device is no longer current after adding particles
    std::cout << "DEBUG: particle_count now = " << particle_count << std::endl;
}

void ReboundCudaSimulation::setGravityMode(GravityMode mode) {
    config.gravity_mode = mode;
}

void ReboundCudaSimulation::setTreeParameters(double opening_angle, double softening) {
    config.opening_angle = opening_angle;
    config.softening = softening;
}

void ReboundCudaSimulation::copyParticlesToDevice() {
    if (!particles_allocated) return;
    
    std::cout << "DEBUG: Copying " << particle_count << " particles to device (config.n_particles=" << config.n_particles << ")" << std::endl;
    
    hipError_t err = hipMemcpy(d_particles, h_particles, 
                                particle_count * sizeof(Particle), hipMemcpyHostToDevice);
    checkCudaError(err, "Failed to copy particles to device");
    
    device_particles_current = true;  // Device is now current
}

void ReboundCudaSimulation::copyParticlesFromDevice() {
    if (!particles_allocated || !device_particles_current) return;
    
    std::cout << "DEBUG: Copying " << particle_count << " particles from device (config.n_particles=" << config.n_particles << ")" << std::endl;
    
    hipError_t err = hipMemcpy(h_particles, d_particles, 
                                particle_count * sizeof(Particle), hipMemcpyDeviceToHost);
    checkCudaError(err, "Failed to copy particles from device");
}

void ReboundCudaSimulation::computeForces() {
    if (particle_count == 0) return;
    
    // Set up kernel launch parameters
    int threadsPerBlock = 256;
    int blocksPerGrid = (particle_count + threadsPerBlock - 1) / threadsPerBlock;
    
    switch (config.gravity_mode) {
        case GRAVITY_NONE:
            // No gravity calculation - just zero out accelerations
            {
                int threadsPerBlock = 256;
                int blocksPerGrid = (particle_count + threadsPerBlock - 1) / threadsPerBlock;
                
                // Zero out only acceleration components, not the entire particle
                dim3 block(threadsPerBlock);
                dim3 grid(blocksPerGrid);
                
                // Simple kernel to zero accelerations (we'll need to add this)
                // For now, just skip gravity calculation
            }
            break;
            
        case GRAVITY_BASIC:
            computeForcesBasicKernel<<<blocksPerGrid, threadsPerBlock>>>(
                d_particles, particle_count, config.G, config.softening);
            break;
            
        case GRAVITY_COMPENSATED:
            computeForcesCompensatedKernel<<<blocksPerGrid, threadsPerBlock>>>(
                d_particles, particle_count, config.G, config.softening);
            break;
            
        case GRAVITY_TREE:
            // Build tree before computing forces
            buildTree();
            computeForcesTreeKernel<<<blocksPerGrid, threadsPerBlock>>>(
                d_particles, oct_tree.getDeviceNodes(), particle_count, 
                config.G, config.opening_angle, config.softening);
            break;
    }
    
    // Check for kernel errors
    hipError_t err = hipGetLastError();
    checkCudaError(err, "Kernel execution failed in computeForces");
    
    // Wait for kernel to complete
    err = hipDeviceSynchronize();
    checkCudaError(err, "Device synchronization failed in computeForces");
}

void ReboundCudaSimulation::updatePositions() {
    if (particle_count == 0) return;
    
    int threadsPerBlock = 256;
    int blocksPerGrid = (particle_count + threadsPerBlock - 1) / threadsPerBlock;
    
    updatePositionsKernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_particles, particle_count, config.dt);
    
    hipError_t err = hipGetLastError();
    checkCudaError(err, "Kernel execution failed in updatePositions");
    
    err = hipDeviceSynchronize();
    checkCudaError(err, "Device synchronization failed in updatePositions");
}

void ReboundCudaSimulation::step() {
    // Leapfrog integration steps:
    
    // 1. Update velocities by half step (kick)
    int threadsPerBlock = 256;
    int blocksPerGrid = (particle_count + threadsPerBlock - 1) / threadsPerBlock;
    
    updateVelocitiesKernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_particles, particle_count, config.dt * 0.5);
    
    // 2. Update positions by full step (drift)
    updatePositions();
    
    // 3. Compute new forces
    computeForces();
    
    // 4. Update velocities by half step (kick)
    updateVelocitiesKernel<<<blocksPerGrid, threadsPerBlock>>>(
        d_particles, particle_count, config.dt * 0.5);
    
    // Update simulation time
    config.t += config.dt;
    
    // Synchronize to ensure all operations are complete
    hipError_t err = hipDeviceSynchronize();
    checkCudaError(err, "Device synchronization failed in step");
}

void ReboundCudaSimulation::integrate(double t_end) {
    std::cout << "DEBUG: Starting integration..." << std::endl;
    
    // Debug: Check particles before copying to device
    std::cout << "DEBUG: Particles before copying to device:" << std::endl;
    for (int i = 0; i < particle_count; i++) {
        Particle& p = h_particles[i];
        std::cout << "  Particle " << i << ": mass=" << p.m << ", pos=(" << p.x << ", " << p.y << ", " << p.z << ")" << std::endl;
    }
    
    // Copy particles to device
    copyParticlesToDevice();
    
    // Debug: Copy back immediately to check if copy worked
    copyParticlesFromDevice();
    std::cout << "DEBUG: Particles after round-trip copy:" << std::endl;
    for (int i = 0; i < particle_count; i++) {
        Particle& p = h_particles[i];
        std::cout << "  Particle " << i << ": mass=" << p.m << ", pos=(" << p.x << ", " << p.y << ", " << p.z << ")" << std::endl;
    }
    
    // Copy to device again for simulation
    copyParticlesToDevice();
    
    // Initial force calculation
    computeForces();
    
    // Debug: Check after force calculation
    copyParticlesFromDevice();
    std::cout << "DEBUG: Particles after force calculation:" << std::endl;
    for (int i = 0; i < particle_count; i++) {
        Particle& p = h_particles[i];
        std::cout << "  Particle " << i << ": mass=" << p.m << ", pos=(" << p.x << ", " << p.y << ", " << p.z << "), acc=(" << p.ax << ", " << p.ay << ", " << p.az << ")" << std::endl;
    }
    
    // Copy back to device for integration
    copyParticlesToDevice();
    
    int steps = 0;
    while (config.t < t_end && steps < config.max_iterations) {
        step();
        steps++;
        
        // Optional: print progress every 1000 steps
        if (steps % 1000 == 0) {
            std::cout << "Step " << steps << ", t = " << config.t << std::endl;
        }
        
        // Early exit for debugging
        if (steps >= 2) break;
    }
    
    // Copy final results back to host
    copyParticlesFromDevice();
}

void ReboundCudaSimulation::printParticles() {
    // If device particles are current (simulation has run), copy from device
    // Otherwise use host particles directly
    if (device_particles_current) {
        copyParticlesFromDevice();
    }
    
    std::cout << "\n=== Particle States ===" << std::endl;
    for (int i = 0; i < particle_count; i++) {
        Particle& p = h_particles[i];
        std::cout << "Particle " << i << ": ";
        std::cout << "pos=(" << p.x << ", " << p.y << ", " << p.z << ") ";
        std::cout << "vel=(" << p.vx << ", " << p.vy << ", " << p.vz << ") ";
        std::cout << "mass=" << p.m << std::endl;
    }
}

double ReboundCudaSimulation::getTotalEnergy() {
    // If device particles are current (simulation has run), copy from device
    // Otherwise use host particles directly
    if (device_particles_current) {
        copyParticlesFromDevice();
    }
    
    double kinetic = 0.0;
    double potential = 0.0;
    
    // Calculate kinetic energy
    for (int i = 0; i < particle_count; i++) {
        Particle& p = h_particles[i];
        double v2 = p.vx*p.vx + p.vy*p.vy + p.vz*p.vz;
        kinetic += 0.5 * p.m * v2;
    }
    
    // Calculate potential energy
    for (int i = 0; i < particle_count; i++) {
        for (int j = i + 1; j < particle_count; j++) {
            Particle& pi = h_particles[i];
            Particle& pj = h_particles[j];
            
            double dx = pj.x - pi.x;
            double dy = pj.y - pi.y;
            double dz = pj.z - pi.z;
            double r = sqrt(dx*dx + dy*dy + dz*dz);
            
            if (r > 1e-15) {
                potential -= config.G * pi.m * pj.m / r;
            }
        }
    }
    
    return kinetic + potential;
}

void ReboundCudaSimulation::buildTree() {
    // Copy particles from device to host for tree construction
    copyParticlesFromDevice();
    
    // Build tree using the OctTree class
    oct_tree.buildTree(h_particles, particle_count);
    
    // Copy tree to device for GPU kernels
    oct_tree.copyToDevice();
} 